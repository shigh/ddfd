#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include "utils.hpp"

/*
 * 2D jacobi iteration
 */
__global__
void jacobi_2d(float *x_d, float *xnew_d, float *b_d,
			   int ny, float dy, int nx, float dx)
{

	int x  = threadIdx.x + blockIdx.x*blockDim.x;
	int y  = threadIdx.y + blockIdx.y*blockDim.y;
	int x0 = x;
	int tid, north, south, east, west;

	float k  =(dx*dx*dy*dy)/(dx*dx + dy*dy);
	float kx = k/(dx*dx);
	float ky = k/(dy*dy);

	while(y < ny)
    {
		while(x < nx)
		{

			tid   = x + y*nx;
			north = tid + nx;
			south = tid - nx;
			west  = tid - 1;
			east  = tid + 1;

			if( x>0 && x<nx-1 && y>0 && y<ny-1)
				xnew_d[tid] = (k*b_d[tid] -
							   ky*(x_d[north] + x_d[south]) -
							   kx*(x_d[west]  + x_d[east]))/(-2.0);

			x += blockDim.x;

		}

		x = x0;
		y += blockDim.y;

    }

}

void call_jacobi_step_2d(thrust::device_vector<float>& x_d,
						 thrust::device_vector<float>& xnew_d,
						 thrust::device_vector<float>& b_d,
						 int ny, float dy, int nx, float dx)
{

	dim3 dB(32, 32);
	dim3 dT(16, 16);

	//call Jacobi step
	jacobi_2d<<<dB, dT>>>(thrust::raw_pointer_cast(&x_d[0]),
						  thrust::raw_pointer_cast(&xnew_d[0]),
						  thrust::raw_pointer_cast(&b_d[0]),
						  ny, dy, nx, dx);
  
}

float jacobi_solve_2d(thrust::device_vector<float>& x_d,
					  thrust::device_vector<float>& b_d,
					  int ny, float dy, int nx, float dx,
					  int max_iter, float tol)
{
   
	thrust::device_vector<float> xnew_d(nx*ny, 0);
	thrust::copy(x_d.begin(), x_d.end(), xnew_d.begin());
	//copy_boundaries(x, xnew, nx, ny);

	// Set to -1 to keep the iteration count correct
	int i = -1;
	// Init error >tol to avoid tripping condition in while
	// before the first iteration
	float error = tol + 1;

	while( error > tol && i < max_iter )
    {

		i++;
      
		//jacobi step
		if( i%2==0 )
			call_jacobi_step_2d(x_d, xnew_d, b_d, ny, dy, nx, dx);
		else
			call_jacobi_step_2d(xnew_d, x_d, b_d, ny, dy, nx, dx);

		//l_infinity norm 
		// error = l_inf_diff(x_d, xnew_d);
		error = two_norm(x_d, xnew_d); 

    }

	if( i%2==0 )
		thrust::copy(xnew_d.begin(), xnew_d.end(), x_d.begin());

	return error;

}

