#include "hip/hip_runtime.h"
/*
 * Test two domain convergence
 */

#include <vector>
#include <cstddef>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <mpi.h>

#include "cusp_poisson.hpp"
#include "solvers.hpp"
#include "test_utils.hpp"
#include "grid.hpp"
#include "utils.hpp"


void make_reference_solution(cusp::array1d<float, cusp::host_memory>& x_full_h,
							 int nz, float dz, int ny, float dy, int nx, float dx)
{

	// Reference solution	
	cusp::array1d<float, cusp::host_memory>   b_h(nz*ny*nx, 0);
	cusp::array1d<float, cusp::device_memory> x_full(nz*ny*nx, 0);

	for(int k=0; k<nz; k++)
		for(int i=0; i<ny; i++)
			for(int j=0; j<nx; j++)
				b_h[j+i*nx+k*nx*ny] = sin(j*dx)*sin(i*dy)*sin(k*dz);

	cusp::array1d<float, cusp::device_memory> b_full(b_h);
	PoissonSolver3DCUSP<float> solver_full(b_full, nz, dz, ny, dy, nx, dx);

	solver_full.solve(x_full);

	x_full_h = cusp::array1d<float, cusp::host_memory>(x_full);

}

void build_b(std::size_t global_nz, float dz,
			 std::size_t global_ny, float dy,
			 std::size_t global_nx, float dx,
			 std::size_t overlap,
			 const std::vector<int>& grid_coords,
			 cusp::array1d<float, cusp::device_memory>& b,
			 std::size_t& nz, std::size_t& ny, std::size_t& nx)
{

	std::vector<std::size_t> start_vec;
	std::vector<std::size_t> end_vec;
	partition_domain(start_vec, end_vec, global_nz, 2, overlap);

	std::size_t x_location = grid_coords[2];
	std::size_t y_location = grid_coords[1];
	std::size_t z_location = grid_coords[0];

	std::size_t x_start = start_vec[x_location];
	std::size_t x_end   = end_vec[x_location];
	nx                  = x_end - x_start;
	std::size_t y_start = 0;//start_vec[y_location];
	std::size_t y_end   = global_ny;//end_vec[y_location];
	ny                  = y_end - y_start;
	std::size_t z_start = 0;//start_vec[z_location];
	std::size_t z_end   = global_nz;//end_vec[z_location];
	nz                  = z_end - z_start;	

	// Local domain
	cusp::array1d<float, cusp::host_memory>   b_h(nx*ny*nz, 0);
	
	std::size_t ind;
	for(std::size_t k=z_start; k<z_end; k++)
		for(std::size_t i=y_start; i<y_end; i++)
			for(std::size_t j=x_start; j<x_end; j++)
			{
				ind = (j-x_start)+(i-y_start)*nx+(k-z_start)*nx*ny;
				b_h[ind] = sin(j*dx)*sin(i*dy)*sin(k*dz);
			}

	b = cusp::array1d<float, cusp::device_memory>(b_h);

}

void build_ref_b(std::size_t global_nz, float dz,
				 std::size_t global_ny, float dy,
				 std::size_t global_nx, float dx,
				 std::size_t overlap,
				 const std::vector<int>& grid_coords,
				 cusp::array1d<float, cusp::device_memory>& x)
{

	std::vector<std::size_t> start_vec;
	std::vector<std::size_t> end_vec;
	partition_domain(start_vec, end_vec, global_nz, 2, overlap);

	std::size_t x_location = grid_coords[2];
	std::size_t y_location = grid_coords[1];
	std::size_t z_location = grid_coords[0];

	std::size_t x_start = start_vec[x_location];
	std::size_t x_end   = end_vec[x_location];
	std::size_t nx      = x_end - x_start;
	std::size_t y_start = 0;//start_vec[y_location];
	std::size_t y_end   = global_ny;//end_vec[y_location];
	std::size_t ny      = y_end - y_start;
	std::size_t z_start = 0;//start_vec[z_location];
	std::size_t z_end   = global_nz;//end_vec[z_location];
	std::size_t nz      = z_end - z_start;	

	cusp::array1d<float, cusp::host_memory> xr;
	make_reference_solution(xr, global_nz, dz, global_ny, dy, global_nx, dx);

	// Local domain
	cusp::array1d<float, cusp::host_memory> x_h(nx*ny*nz, 0);
	
	std::size_t ind;
	for(std::size_t k=z_start; k<z_end; k++)
		for(std::size_t i=y_start; i<y_end; i++)
			for(std::size_t j=x_start; j<x_end; j++)
			{
				ind = (j-x_start)+(i-y_start)*nx+(k-z_start)*nx*ny;
				x_h[ind] = xr[j+i*nx+k*nx*ny];
			}

	x = cusp::array1d<float, cusp::device_memory>(x_h);

}


void poisson3d(MPI_Comm cart_comm,
			   std::vector<int> grid_dim,
			   cusp::array1d<float, cusp::device_memory> x,
			   cusp::array1d<float, cusp::device_memory> b,
			   std::size_t nz, float dz,
			   std::size_t ny, float dy,
			   std::size_t nx, float dx,
			   std::size_t overlap)
{


	int grid_rank;
	std::vector<int> grid_coords(3);
	MPI_Comm_rank(cart_comm, &grid_rank);
	MPI_Cart_coords(cart_comm, grid_rank, 3, &grid_coords[0]);

	bool has_east = grid_coords[2] < grid_dim[2]-1;
	bool has_west = grid_coords[2] > 0;

	std::vector<int> tmp_coords(3);		
	int east = -1;
	if(has_east)
	{
		tmp_coords = grid_coords;
		tmp_coords[2] += 1;
		MPI_Cart_rank(cart_comm, &tmp_coords[0], &east);
	}
	int west = -1;
	if(has_west)
	{
		tmp_coords = grid_coords;
		tmp_coords[2] -= 1;
		MPI_Cart_rank(cart_comm, &tmp_coords[0], &west);
	}


	PoissonSolver3DCUSP<float> solver(b, nz, dz, ny, dy, nx, dx);

	DeviceBoundarySet<float> device_bs(nz, ny, nx);
	HostBoundarySet<float> host_bs(nz, ny, nx);
	HostBoundarySet<float> host_bs_r(nz, ny, nx);

	thrust::device_vector<float> tmp(nx*ny*nz, 0);


	const int n_iter = 10;
	for(int i=0; i<n_iter; i++)
	{

		std::cout << i << std::endl;

	    solver.solve(x);

		extract_all_boundaries(thrust::raw_pointer_cast(&x[0]), device_bs,
							   nz, ny, nx, overlap);
		host_bs.copy(device_bs);

		hipDeviceSynchronize();

		MPI_Status east_status, west_status;

		if(has_east)
			MPI_Send(host_bs.get_east_ptr(), ny*nz, MPI_FLOAT, east, 0, cart_comm);
		if(has_east)
			MPI_Recv(host_bs_r.get_east_ptr(), ny*nz, MPI_FLOAT, east, 0, cart_comm, &east_status);
		
		if(has_west)
			MPI_Send(host_bs.get_west_ptr(), ny*nz, MPI_FLOAT, west, 0, cart_comm);
		if(has_west)
			MPI_Recv(host_bs_r.get_west_ptr(), ny*nz, MPI_FLOAT, west, 0, cart_comm, &west_status);

		device_bs.copy(host_bs_r);		

		if(has_east)
			set_east<float>(device_bs.get_east_ptr(), thrust::raw_pointer_cast(&x[0]),
							nz, ny, nx);

		if(has_west)
			set_west<float>(device_bs.get_west_ptr(), thrust::raw_pointer_cast(&x[0]),
							nz, ny, nx);


	}


}
			   
			   

int main(int argc, char* argv[])
{

	int size, rank;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &size);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	
	std::vector<int> dimensions(3, 1);
	std::vector<int> wrap_around(3, 0);
	dimensions[0] = 1; dimensions[1] = 1; dimensions[2] = size;

	MPI_Comm cart_comm;
	MPI_Cart_create(MPI_COMM_WORLD, 3, &dimensions[0],
					&wrap_around[0], 1, &cart_comm);


	int grid_rank;
	std::vector<int> grid_coords(3);
	MPI_Comm_rank(cart_comm, &grid_rank);
	MPI_Cart_coords(cart_comm, grid_rank, 3, &grid_coords[0]);


	std::size_t global_nx = 10;
	std::size_t global_ny = global_nx;
	std::size_t global_nz = global_ny;

	float dx = 2*M_PI/(global_nx-1.);
	float dy = 2*M_PI/(global_ny-1.);
	float dz = 2*M_PI/(global_nz-1.);

	std::size_t overlap = 2;

	std::size_t nz, ny, nx;
	cusp::array1d<float, cusp::device_memory> b;
	build_b(global_nz, dz, global_ny, dy, global_nx, dx,
			overlap, grid_coords, b,
			nz, ny, nx);

	cusp::array1d<float, cusp::device_memory> x(nx*ny*nz, 0);
	
	poisson3d(cart_comm, dimensions, x, b,
			  nz, dz, ny, dy, nx, dx, overlap);


	cusp::array1d<float, cusp::device_memory> xr;
	build_ref_b(global_nz, dz, global_ny, dy,
				global_nx, dx,
				overlap, grid_coords, xr);

	float error = 0;
	for(int i=0; i<xr.size(); i++)
		error = max(error, (xr[i] - x[i])*(xr[i] - x[i]));

	std::cout << error << std::endl;

	if(rank==0)
	{		
		save_vector(x, "of.txt");
		save_vector(xr, "ofb.txt");
	}

	MPI_Finalize();

	return 0;

}

