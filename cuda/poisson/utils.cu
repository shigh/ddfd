#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <stdio.h>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>


struct AbsDiff
{
  __host__ __device__
  float operator()(float a, float b)
  {
    return abs(a-b);
  }

};

float l_inf_diff(thrust::device_vector<float>& a,
 	         thrust::device_vector<float>& b)
{
  thrust::device_vector<float> tmp(a.size());
  thrust::transform(a.begin(), a.end(), b.begin(),
		   tmp.begin(), AbsDiff());

  thrust::device_vector<float>::iterator max_it =
    thrust::max_element(tmp.begin(), tmp.end());
  
  return *max_it;
}

/*
 * Copy boundaries from A into B
 */
void copy_boundaries(thrust::device_vector<float>& A,
		     thrust::device_vector<float>& B,
		     int ny, int nx)
{

  // First and last rows
  thrust::copy_n(A.begin(), nx, B.begin());
  thrust::copy_n(&A[(ny-1)*nx], nx, &B[(ny-1)*nx]);

  // First and last cols
  for(int i=0; i<ny; i++)
    {
      B[i*nx]      = A[i*nx];
      B[i*nx+nx-1] = A[i*nx+nx-1];
    }

}
